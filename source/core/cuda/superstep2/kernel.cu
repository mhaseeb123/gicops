#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2022 Muhammad Haseeb, and Fahad Saeed
 * Florida International University, Miami, FL
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <https://www.gnu.org/licenses/>.
 *
 */

#include <hip/hip_runtime.h>
#include <string>
#include <thrust/gather.h>
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/sequence.h>
#include <thrust/distance.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>

#include <iostream>
#include <thread>

#include "cuda/driver.hpp"
#include "cuda/superstep2/kernel.hpp"

using namespace std;

extern gParams params;

// GPU thread block size
const int MAXBLOCK = 1024;
const int TEMPVECTOR_SIZE = KBYTES(16);
const int BATCHSIZE = 20000;


// -------------------------------------------------------------------------------------------- //

namespace hcp 
{

namespace gpu
{

namespace cuda
{

namespace s2
{

// -------------------------------------------------------------------------------------------- //

//
// preprocess
//
void preprocess(MSQuery *query, string_t &filename, int fileindex)
{
    // Read and preprocess the input MS2 data.
    if (params.filetype == gParams::FileType_t::PBIN)
    {
        // local variables
        int maxlen = 0;
        int nqueries = 0;

        // read MS2 file, preprocess and write data to disk
        auto vals = readAndPreprocess(filename);

        nqueries = vals[0];
        maxlen = vals[1];

        // compute number of chunks
        int nchunks = (nqueries / QCHUNK) + (nqueries % QCHUNK > 0);

        // new filename with .bin extension
        string_t filename_bin(std::move(filename + ".pbin"));

        // set the filename
        query->setFilename(filename_bin);

        // initialize the MSQuery::info_t
        query->Info() = info_t(maxlen, nchunks, nqueries);
    }
    else
    {
        // no GPU processing needed here. simply
        // call the query->initialize function
        query->initialize(&filename, fileindex);
    }
}

// -------------------------------------------------------------------------------------------- //


//
// GPU kernel for picking peaks
//
__global__ void pickpeaks(spectype_t *d_intns, spectype_t * d_mzs, spectype_t *d_m_intns, spectype_t *d_m_mzs, int *d_lens, int *d_m_lens)
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int bsize = blockDim.x;

    int newspectrumsize = d_m_lens[bid+1] - d_m_lens[bid];

    auto src_intns = d_intns + d_lens[bid+1] - newspectrumsize;
    auto src_mzs = d_mzs + d_lens[bid+1] - newspectrumsize;

    auto dst_intns = d_m_intns + d_m_lens[bid];
    auto dst_mzs = d_m_mzs + d_m_lens[bid];

    for (; tid < newspectrumsize; tid+=bsize)
    {
        dst_intns[tid] = src_intns[tid];
        dst_mzs[tid] = src_mzs[tid];
    }
}

// -------------------------------------------------------------------------------------------- //

//
// kernel to preprocess the data, reduce and compute new spectrum lengths
//
__global__ void computenewlens(spectype_t *d_intns, int *d_lens, int *d_m_lens, int base_int, int min_int)
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;

    int  warpsize = 32;
    int  warpId = tid / warpsize;
    int laneId = tid % warpsize;

    // how many elements in the current array
    int startidx = d_lens[bid];
    int endidx = d_lens[bid+1];

    int SpectrumSize = endidx - startidx;
    int maxelements = MIN(QALEN, SpectrumSize);

    spectype_t *loc_intns = d_intns + endidx - maxelements;

    double factor = ((double_t) base_int) / loc_intns[maxelements - 1];

    // filter out intensities > params.min_int (or 1% of base peak)
    auto l_min_int = min_int;

    int myVal = 0;

    // normalize intensities
    if (tid < maxelements)
    {
        loc_intns[tid] *= factor;
        myVal = (loc_intns[tid] >= l_min_int) ? 1 : 0;
    }

    __syncthreads();

    unsigned mask  = __ballot_sync(0xffffffff, tid < maxelements);

    for(int offset = warpsize / 2; offset > 0; offset /= 2)
    {
        int tempVal = __shfl_down_sync(mask, myVal, offset);
        myVal += tempVal;
    }

    __syncthreads();

    __shared__ int locTots[32];

    if (laneId == 0)
    {
        locTots[warpId] = myVal;
    }

    __syncthreads();

    int nwarps = maxelements / warpsize + (maxelements % warpsize > 0);

    if (tid < nwarps)
        myVal = locTots[tid];
    else
        myVal = 0;

    __syncthreads();

    if (warpId == 0)
    {
        unsigned int mask  = __ballot_sync(0xffffffff, tid < nwarps);

        for(int offset = warpsize / 2; offset > 0; offset /= 2)
        {
            int tempVal = __shfl_down_sync(mask, myVal, offset);
            myVal += tempVal;
        }
    }

    __syncthreads();

    // the final value should be at location zero
    if (tid == 0)
        d_m_lens[bid] = myVal;
}

// -------------------------------------------------------------------------------------------- //

//
// kernel to generate array indices
//
template <typename T>
__global__ void generateArrayNums(int N, int * d_lens, T *d_arraynums, spectype_t *d_tmp)
{
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int bsize = blockDim.x;

    // how many elements in the current array
    int startidx = d_lens[bid];
    int endidx = d_lens[bid+1];

    // write the array numbers to d_arraynums
    for (int i = startidx + tid; i < endidx; i += bsize)
    {
        d_arraynums[i] = bid;
        d_tmp[i] = bid;
    }
}

// -------------------------------------------------------------------------------------------- //

//
// The GPU-ArraySort kernel
//
status_t ArraySort(spectype_t *intns, spectype_t *mzs, int *lens, int &idx, int count, int maxslen, spectype_t *m_intn, spectype_t *m_mzs)
{
    // get driver object
    auto driver = hcp::gpu::cuda::driver::get_instance();

    // device ptrs for raw and processed data and lengths
    spectype_t * d_intns = nullptr;
    spectype_t * d_mzs = nullptr;
    int *d_arraynums = nullptr;
    int *d_indices = nullptr;

    int *d_lens = nullptr;
    int *d_m_lens = nullptr;

    // output vectors
    spectype_t *d_m_intns = nullptr;
    spectype_t *d_m_mzs = nullptr;

    // the raw buffer size
    int rawsize = idx;

    // temporary vector for data gathering
    spectype_t *d_tmp = nullptr;
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_allocate_async(d_tmp, rawsize, driver->stream[3])); //BATCHSIZE * TEMPVECTOR_SIZE));

    // -------------------------------------------------------------------------------------------- //

    //
    // allocate device memory and transfer data
    //

    // memory for intensities
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_allocate_async(d_intns, rawsize, driver->stream[0]));

    // transfer intensities to the GPU
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::H2D(d_intns, intns, idx, driver->stream[0]));

    // memory for m/zs
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_allocate_async(d_mzs, rawsize, driver->stream[1]));

    // transfer m/zs to the GPU
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::H2D(d_mzs, mzs, idx, driver->stream[1]));

    // memory for spectrum lengths
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_allocate_async(d_lens, BATCHSIZE+1, driver->stream[2]));

    // transfer lengths to the GPU
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::H2D(d_lens, lens, count, driver->stream[2]));

    // memory for arraynums
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_allocate_async(d_arraynums, rawsize, driver->stream[2])); 

    // memory for indices
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_allocate_async(d_indices, rawsize, driver->stream[3])); 

    // memory for processed intensities
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_allocate_async(d_m_intns, QALEN * BATCHSIZE, driver->stream[3]));

    // memory for processed mzs
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_allocate_async(d_m_mzs, QALEN * BATCHSIZE, driver->stream[3]));

    // memory for new spectrum lengths
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_allocate_async(d_m_lens, BATCHSIZE+1, driver->stream[3]));

    // -------------------------------------------------------------------------------------------- //

    //
    // initialize sequences and arraynums
    //

    // initialize indices to sequences - independent of device data
    thrust::sequence(thrust::device.on(driver->get_stream(3)), d_indices, d_indices + idx);

    // compute an exclusive scan of the lengths
    thrust::exclusive_scan(thrust::device.on(driver->get_stream(2)), d_lens, d_lens + count + 1, d_lens, 0);

    // maxlen cannot be larger than the largest allowed blocksize on CUDA
    auto maxlen = MIN(MAXBLOCK, maxslen);

    // generate array numbers.
    // NOTE 2 SELF: DEPENDS on exclusive_scan(d_lens) - do not move it up
    hcp::gpu::cuda::s2::generateArrayNums<<<count, maxlen, 48, driver->get_stream(2)>>>(idx, d_lens, d_arraynums, d_tmp);

    // synchronize data transfers before calling the kernels
    driver->all_streams_sync();

    // -------------------------------------------------------------------------------------------- //

    //
    // the GPU-ArraySort algorithm
    //

    // stable sort by key
    thrust::stable_sort_by_key(thrust::device.on(driver->get_stream()), d_intns, d_intns + idx, d_indices);
    thrust::gather(thrust::device.on(driver->get_stream()), d_indices, d_indices + idx, d_mzs, d_tmp);
    thrust::gather(thrust::device.on(driver->get_stream()), d_indices, d_indices + idx, d_arraynums, d_mzs);

    // reinitialize indices
    thrust::sequence(thrust::device.on(driver->get_stream()), d_indices, d_indices + idx);

    // stable sort by key
    thrust::stable_sort_by_key(thrust::device.on(driver->get_stream()), d_mzs, d_mzs + idx, d_indices);
    thrust::gather(thrust::device.on(driver->get_stream()), d_indices, d_indices + idx, d_tmp, d_mzs);
    thrust::gather(thrust::device.on(driver->get_stream()), d_indices, d_indices + idx, d_intns, d_tmp);

    // synchronize before doing the swaps
    driver->stream_sync();

    // swap d_intns and d_tmp
    auto d_swap = d_tmp;
    d_tmp = d_intns;
    d_intns = d_swap;

    // not really needed but for completeness
    int shmem = QALEN * sizeof(short);

    // -------------------------------------------------------------------------------------------- //

    //
    // normalize the data, pick top 100 and compute new spectrum lengths
    //

    // pickpeaks in the output array
    hcp::gpu::cuda::s2::computenewlens<<<count, QALEN, shmem, driver->get_stream()>>>(d_intns, d_lens, d_m_lens, params.base_int, params.min_int);

    // -------------------------------------------------------------------------------------------- //

    // transfer processed lengths back to the CPU
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::D2H(lens, d_m_lens, count, driver->stream[0]));

    // -------------------------------------------------------------------------------------------- //

    //
    // compute an exclusive scan of the new lengths and copy to the output arrays
    //

    thrust::exclusive_scan(thrust::device.on(driver->get_stream()), d_m_lens, d_m_lens + count + 1, d_m_lens, 0);

    // pick the peaks
    hcp::gpu::cuda::s2::pickpeaks<<<count, QALEN, shmem, driver->get_stream()>>>(d_intns, d_mzs, d_m_intns, d_m_mzs, d_lens, d_m_lens);

    // -------------------------------------------------------------------------------------------- //

    // get new total length (idx)

    int *h_idxptr = &idx;
    int *d_idxptr = d_m_lens + count;
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::D2H(h_idxptr, d_idxptr, 1, driver->stream[0]));

    // this is critical for the next step
    driver->stream_sync();

    // -------------------------------------------------------------------------------------------- //

    //
    // transfer processed data back to CPU
    //

    hcp::gpu::cuda::error_check(hcp::gpu::cuda::D2H(m_intn, d_m_intns, idx, driver->stream[0]));
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::D2H(m_mzs, d_m_mzs, idx, driver->stream[0]));

    // synchronize all streams before deallocating device memory
    driver->all_streams_sync();

    // -------------------------------------------------------------------------------------------- //

    // free the GPU memory

    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_free_async(d_mzs, driver->stream[0]));
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_free_async(d_intns, driver->stream[0]));
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_free_async(d_arraynums, driver->stream[0]));
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_free_async(d_indices, driver->stream[0]));
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_free_async(d_tmp, driver->stream[0]));
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_free_async(d_m_mzs, driver->stream[0]));
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_free_async(d_m_intns, driver->stream[0]));
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_free_async(d_lens, driver->stream[0]));
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_free_async(d_m_lens, driver->stream[0]));


    // return success status
    return SLM_SUCCESS;
}

// -------------------------------------------------------------------------------------------- //

//
// readAndPreprocess spectra
//
std::array<int, 2> readAndPreprocess(string_t &filename)
{
    int_t largestspec = 0;
    int_t largestspec_loc = 0;
    int_t count = 0;
    int_t globalcount = 0;
    int_t specsize = 0;
    int_t m_idx = 0;

    char_t *Zsave;
    char_t *Isave;

    // host data vectors
    spectype_t * mzs = nullptr;
    spectype_t *intns = nullptr;
    int *lens = nullptr;

    // reverse BATCHSIZE x (20 * 1024) host vectors
    hcp::gpu::cuda::host_pinned_allocate<spectype_t>(mzs, TEMPVECTOR_SIZE * BATCHSIZE);
    hcp::gpu::cuda::host_pinned_allocate<spectype_t>(intns, TEMPVECTOR_SIZE * BATCHSIZE);
    hcp::gpu::cuda::host_pinned_allocate<int>(lens, BATCHSIZE);

    // output data
    spectype_t *m_mzs = nullptr;
    spectype_t *m_intns = nullptr;

    hcp::gpu::cuda::host_pinned_allocate(m_mzs, BATCHSIZE * QALEN);
    hcp::gpu::cuda::host_pinned_allocate(m_intns, BATCHSIZE * QALEN);

    float *rtimes = new float[2 * BATCHSIZE];
    float *prec_mz = rtimes + BATCHSIZE;
    int *z = new int[BATCHSIZE];

    /* Get a new ifstream object and open file */
    ifstream *qqfile = new ifstream(filename);

    /* Check if file opened */
    if (qqfile->is_open())
    {
        string_t line;
        bool isFirst = true;

        /* While we still have lines in MS2 file */
        while (!qqfile->eof())
        {
            /* Read one line */
            getline(*qqfile, line);

            if (line.empty() || line[0] == 'H' || line[0] == 'D')
            {
                continue;
            }
            /* Scan: (S) */
            else if (line[0] == 'S')
            {
                if (!isFirst)
                {
                    largestspec = max(specsize, largestspec);
                    largestspec_loc = max(specsize, largestspec_loc);
                    // write the updated specsize
                    lens[count] = specsize;
                    //m_idx += specsize;

                    count++;
                    globalcount++;

                    // if the buffer is full, then dump to file
                    if (count == BATCHSIZE)
                    {
                        // lens will update after this
                        hcp::gpu::cuda::s2::ArraySort(intns, mzs, lens, m_idx, count, largestspec_loc, m_intns, m_mzs);

                        // flush to the binary file
                        MSQuery::flushBinaryFile(&filename, m_mzs, m_intns, rtimes, prec_mz, z, lens, count);
                        
                        count = 0;
                        m_idx = 0;
                        largestspec_loc = 0;
                    }
                }
                else
                    isFirst = false;

                specsize = 0;

            }
            else if (line[0] == 'Z')
            {
                char_t *mh = strtok_r((char_t *) line.c_str(), " \t", &Zsave);
                mh = strtok_r(NULL, " \t", &Zsave);
                string_t val = "1";

                if (mh != NULL)
                    val = string_t(mh);

                z[count] = MAX(1, std::atoi(val.c_str()));

                val = "0.01";
                mh = strtok_r(NULL, " \t", &Zsave);

                if (mh != NULL)
                    val = string_t(mh);

                prec_mz[count] = std::atof(val.c_str());
            }
            else if (line[0] == 'I')
            {
                char_t *mh = strtok_r((char_t *) line.c_str(), " \t", &Isave);
                mh = strtok_r(NULL, " \t", &Isave);
                string_t val = "";

                if (mh != NULL)
                {
                    val = string_t(mh);
                }

                if (val.compare("RTime") == 0)
                {
                    val = "0.00";
                    mh = strtok_r(NULL, " \t", &Isave);

                    if (mh != NULL)
                    {
                        val = string_t(mh);
                    }

                    rtimes[count] = MAX(0.0, std::atof(val.c_str()));
                }
            }
            /* MS/MS data: [m/z] [int] */
            else
            {
                /* Split line into two DOUBLEs
                 * using space as delimiter */

                char_t *mz1 = strtok_r((char_t *) line.c_str(), " ", &Zsave);
                char_t *intn1 = strtok_r(NULL, " ", &Zsave);
                string_t mz = "0.01";
                string_t intn = "0.01";

                if (mz1 != NULL)
                {
                    mz = string_t(mz1);
                }

                if (intn1 != NULL)
                {
                    intn = string_t(intn1);
                }

                // integrize the values if spectype_t is int
                if constexpr (std::is_same<int, spectype_t>::value)
                {
                    mzs[m_idx] = std::atof(mz.c_str()) * params.scale;
                    intns[m_idx] = std::atof(intn.c_str()) * YAXISMULTIPLIER;
                }
                else
                {
                    mzs[m_idx] = std::atof(mz.c_str());
                    intns[m_idx] = std::atof(intn.c_str());
                }

                // increment the spectrum size & m_idx (cumulative spectrum size)
                specsize++;
                m_idx++;
            }
        }

        largestspec = max(specsize, largestspec);
        largestspec_loc = max(specsize, largestspec_loc);


        lens[count] = specsize;
        // m_idx += specsize;

        count++;
        globalcount++;

        // lens will update after this
        hcp::gpu::cuda::s2::ArraySort(intns, mzs, lens, m_idx, count, largestspec_loc, m_intns, m_mzs);

        // flush the last batch to the binary file
        MSQuery::flushBinaryFile(&filename, m_mzs, m_intns, rtimes, prec_mz, z, lens, count, true);

        // no need to reset count and m_idx here

        /* Close the file */
        qqfile->close();

        delete qqfile;
    }
    else
        std::cout << "Error: Unable to open file: " << filename << std::endl;

    largestspec = max(specsize, largestspec);

    hcp::gpu::cuda::host_pinned_free(intns);
    hcp::gpu::cuda::host_pinned_free(mzs);
    hcp::gpu::cuda::host_pinned_free(lens);
    hcp::gpu::cuda::host_pinned_free(m_intns);
    hcp::gpu::cuda::host_pinned_free(m_mzs);

    // delete the temp arrays
    delete[] rtimes;
    delete[] z;

    // return global count and largest spectrum length
    return std::array<int, 2>{globalcount, largestspec};
}

// -------------------------------------------------------------------------------------------- //

} // namespace s2

} // namespace cuda

} // namespace gpu

} // namespace hcp