#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2022 Muhammad Haseeb, and Fahad Saeed
 * Florida International University, Miami, FL
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <https://www.gnu.org/licenses/>.
 *
 */

#include <hip/hip_runtime.h>

#include <string>
#include <iostream>
#include <thread>

#include "dslim_fileout.h"

#include "cuda/driver.hpp"
#include "cuda/superstep4/kernel.hpp"

// FIXME

#ifdef USE_MPI

#include "dslim.h"

#endif // USE_MPI

using namespace std;

// host side global parameters
extern gParams params;

// -------------------------------------------------------------------------------------------- //

namespace hcp 
{

namespace gpu
{

namespace cuda
{

namespace s4
{

// gumbel curve fitting
__global__ void logWeibullFit(dScores_t *d_Scores, double *evalues, short min_cpsm);

// tail fit method
__global__ void TailFit(dScores_t *d_Scores, double *evalues);

// alternate tail fit method
__global__ void TailFit(double_t *data, float_t *hyp, int *cpsms, double *evalues);

template <class T>
__device__ void LinearFit(T* x, T* y, int_t n, double_t *a, double_t *b);

template <typename T>
__device__ void argmax(T *data, short i1, short i2, T val, short &out);

template <typename T>
__device__ void largmax(T *data, short i1, short i2, T val, short &out);

template <typename T>
__device__ void rargmax(T *data, short i1, short i2, T val, short &out);

template <typename T>
extern __device__ void Assign(T *p_x, T *beg, T *end);

template <typename T>
extern __device__ void partialSum(T *beg, T *end, T *out);

template <typename T>
extern __device__ void XYbar(T *x, T *y, int n, double &xbar, double &ybar);

template <typename T>
__device__ void TopBot(T *x, T *y, int n, const double xbar, const double ybar, double &top, double &bot);

// -------------------------------------------------------------------------------------------- //

__host__ double *& getd_eValues()
{
    static auto driver = hcp::gpu::cuda::driver::get_instance();

    static thread_local double *d_evalues = nullptr;

    if (!d_evalues)
        hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_allocate_async<double>(d_evalues, QCHUNK, driver->stream[DATA_STREAM]));

    return d_evalues;
}

// -------------------------------------------------------------------------------------------- //

__host__ void freed_eValues()
{
    auto driver = hcp::gpu::cuda::driver::get_instance();

    auto &&d_evalues = getd_eValues();
    
    if (d_evalues)
    {
        hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_free_async(d_evalues, driver->stream[DATA_STREAM]));
        d_evalues = nullptr;
    }
}

// -------------------------------------------------------------------------------------------- //

// tail fit method
__global__ void TailFit(dScores_t *d_Scores, double *evalues)
{
    // each block will process one result
    int tid = threadIdx.x;
    int bid = blockIdx.x;

    /* Assign to internal variables */
    auto yy = d_Scores[bid].survival;

    // make sure the hyp is scaled to int here
    int hyp = (d_Scores[bid].topscore->hyperscore * 10 + 0.5);

    double p_x_mem[HISTOGRAM_SIZE];
    double s_x_mem[HISTOGRAM_SIZE];
    double x_axis_mem[HISTOGRAM_SIZE];

    double *p_x = &p_x_mem[0];
    double *sx = &s_x_mem[0];
    double *X = &x_axis_mem[0];

    double mu = 0.0;
    double beta = 4.0;

    short stt1 = 0;
    short stt =  0;
    short end1 = HISTOGRAM_SIZE - 1;
    //short ends = HISTOGRAM_SIZE - 1;

    /* Find the curve region */
    hcp::gpu::cuda::s4::rargmax<double_t>(yy, 0, hyp-1, 1.0, end1);
    hcp::gpu::cuda::s4::argmax<double_t>(yy, 0, end1, 1.0, stt1);

    /* To handle special cases */
    if (stt1 == end1)
    {
        stt1 = end1;
        end1 += 1;
    }

    /* Slice off yyt between stt1 and end1 */
    hcp::gpu::cuda::s4::Assign<double>(p_x, yy + stt1, yy + end1 + 1);

    /* Database size
     * vaa = accumulate(yy, yy + hyp + 1, 0); */
    int vaa = d_Scores[bid].cpsms[tid];

    /* Check if no distribution data except for hyp */
    if (vaa < 1)
    {
        mu = 0;
        beta = 100;
        stt = stt1;
        //ends = end1;
    }
    else
    {
        /* Filter p_x again */
        //ends = end1;
        stt = stt1;

        int p_x_size = end1 - stt1 + 1;

        /* Compute survival function s(x) */
        hcp::gpu::cuda::s4::Assign(sx, p_x, p_x + p_x_size);

        /* cumulative_sum(sx) */
        hcp::gpu::cuda::s4::partialSum(p_x, p_x + p_x_size, sx);

        /* Adjust for negatives */
        short replacement = 0;

        short sx_size = end1 - stt1 + 1;
        short sx_size_1 = sx_size - 1;
        hcp::gpu::cuda::s4::rargmax<double_t>(sx, (short)0, sx_size_1, (double)(1e-4), replacement);

        /* Survival function s(x) */
        for (int j = tid; j < end1 - stt1 + 1; j+=blockDim.x)
        {
            // divide by vaa
            sx[j] /= vaa;
            // s(x) = -(s(x) - 1) = 1 - s(x)
            sx[j] = 1 - sx[j];

            // take care of the case where s(x) > 1
            if (sx[j] > 1)
                sx[j] = 0.999;
            // take care of the case where s(x) < 0
            else if (sx[j] < 0)
                sx[j] = sx[replacement];

            // log10(s(x))
            sx[j] = log(sx[j]);
        }

        __syncthreads();

        /* Offset markers */
        short mark = 0;
        short mark2 = 0;
        auto hgt = sx[sx_size - 1] - sx[0];

        /* If length > 4, then find thresholds */
        if (sx_size > 3)
        {
            hcp::gpu::cuda::s4::largmax<double_t>(sx, 0, sx_size-1, sx[0] + hgt * 0.22, mark);
            mark -= 1;
            hcp::gpu::cuda::s4::rargmax<double_t>(sx, 0, sx_size-1, sx[0] + hgt*0.87, mark2);

            if (mark2 == sx_size)
            {
                mark2 -= 1;
            }

            /* To handle special cases */
            if (mark >= mark2)
            {
                mark = mark2 - 1;
            }
        }
        /* If length < 4 business as usual */
        else if (sx_size == 3)
        {
            /* Mark the start of the regression point */
            hcp::gpu::cuda::s4::largmax<double_t>(sx, 0, sx_size-1, (sx[0] + hgt * 0.22), mark);
            mark -= 1;
            mark2 = sx_size - 1;

            /* To handle special cases */
            if (mark >= mark2)
            {
                mark = mark2 - 1;
            }
        }
        else
        {
            mark = 0;
            mark2 = sx_size - 1;
        }

        __syncthreads();


        for (int jj = stt + tid; jj <= stt + mark2; jj+=blockDim.x)
            // X->AddRange(mark, mark2);
            X[jj - stt] = jj;

        __syncthreads();

        for (int jj = tid; jj <= (mark2-mark); jj+=blockDim.x)
            // sx->clip(mark, mark2);
            sx[jj] = sx[jj+mark];

        __syncthreads();

        hcp::gpu::cuda::s4::LinearFit<double_t>(X, sx, sx_size, &mu, &beta);

        //std::cout << "y = " << mu_t << "x + " << beta_t << std::endl;
        //std::cout << "eValue: " << pow(10, hyp * mu_t + beta_t) * vaa << std::endl;
    }

    /* Estimate the log(s(x)) */
    double_t lgs_x = (mu * hyp) + beta;

    /* Compute the e(x) = n * s(x) = n * 10^(lg(s(x))) */
    evalues[bid] = d_Scores[bid].cpsms[tid] * pow(10.0, lgs_x);

}

// -------------------------------------------------------------------------------------------- //

// tail fit method
__global__ void TailFit(double_t *data, float *hyps, int *cpsms, double *evalues)
{
    // each block will process one result
    int tid = threadIdx.x;
    int bid = blockIdx.x;

    /* Assign to internal variables */
    auto yy = &data[HISTOGRAM_SIZE * bid];

    // make sure the hyp is scaled to int here
    int hyp = hyps[bid] * 10 + 0.5;

    double p_x_mem[HISTOGRAM_SIZE];
    double s_x_mem[HISTOGRAM_SIZE];
    double x_axis_mem[HISTOGRAM_SIZE];

    double *p_x = &p_x_mem[0];
    double *sx = &s_x_mem[0];
    double *X = &x_axis_mem[0];

    double mu = 0.0;
    double beta = 4.0;

    short stt1 = 0;
    short stt =  0;
    short end1 = HISTOGRAM_SIZE - 1;
    //short ends = HISTOGRAM_SIZE - 1;

    /* Find the curve region */
    hcp::gpu::cuda::s4::rargmax<double_t>(yy, 0, hyp-1, 1.0, end1);
    hcp::gpu::cuda::s4::argmax<double_t>(yy, 0, end1, 1.0, stt1);

    /* To handle special cases */
    if (stt1 == end1)
    {
        stt1 = end1;
        end1 += 1;
    }

    /* Slice off yyt between stt1 and end1 */
    hcp::gpu::cuda::s4::Assign<double>(p_x, yy + stt1, yy + end1 + 1);

    /* Database size
     * vaa = accumulate(yy, yy + hyp + 1, 0); */
    int vaa = cpsms[bid];

    /* Check if no distribution data except for hyp */
    if (vaa < 1)
    {
        mu = 0;
        beta = 100;
        stt = stt1;
        //ends = end1;
    }
    else
    {
        /* Filter p_x again */
        //ends = end1;
        stt = stt1;

        int p_x_size = end1 - stt1 + 1;

        /* Compute survival function s(x) */
        hcp::gpu::cuda::s4::Assign(sx, p_x, p_x + p_x_size);

        /* cumulative_sum(sx) */
        hcp::gpu::cuda::s4::partialSum(p_x, p_x + p_x_size, sx);

        /* Adjust for negatives */
        short replacement = 0;

        short sx_size = end1 - stt1 + 1;
        short sx_size_1 = sx_size - 1;
        hcp::gpu::cuda::s4::rargmax<double_t>(sx, (short)0, sx_size_1, (double)(1e-4), replacement);

        /* Survival function s(x) */
        for (int j = tid; j < end1 - stt1 + 1; j+=blockDim.x)
        {
            // divide by vaa
            sx[j] /= vaa;
            // s(x) = -(s(x) - 1) = 1 - s(x)
            sx[j] = 1 - sx[j];

            // take care of the case where s(x) > 1
            if (sx[j] > 1)
                sx[j] = 0.999;
            // take care of the case where s(x) < 0
            else if (sx[j] < 0)
                sx[j] = sx[replacement];

            // log10(s(x))
            sx[j] = log(sx[j]);
        }

        __syncthreads();

        /* Offset markers */
        short mark = 0;
        short mark2 = 0;
        auto hgt = sx[sx_size - 1] - sx[0];

        /* If length > 4, then find thresholds */
        if (sx_size > 3)
        {
            hcp::gpu::cuda::s4::largmax<double_t>(sx, 0, sx_size-1, sx[0] + hgt * 0.22, mark);
            mark -= 1;
            hcp::gpu::cuda::s4::rargmax<double_t>(sx, 0, sx_size-1, sx[0] + hgt*0.87, mark2);

            if (mark2 == sx_size)
            {
                mark2 -= 1;
            }

            /* To handle special cases */
            if (mark >= mark2)
            {
                mark = mark2 - 1;
            }
        }
        /* If length < 4 business as usual */
        else if (sx_size == 3)
        {
            /* Mark the start of the regression point */
            hcp::gpu::cuda::s4::largmax<double_t>(sx, 0, sx_size-1, (sx[0] + hgt * 0.22), mark);
            mark -= 1;
            mark2 = sx_size - 1;

            /* To handle special cases */
            if (mark >= mark2)
            {
                mark = mark2 - 1;
            }
        }
        else
        {
            mark = 0;
            mark2 = sx_size - 1;
        }

        __syncthreads();


        for (int jj = stt + tid; jj <= stt + mark2; jj+=blockDim.x)
            // X->AddRange(mark, mark2);
            X[jj - stt] = jj;

        __syncthreads();

        for (int jj = tid; jj <= (mark2-mark); jj+=blockDim.x)
            // sx->clip(mark, mark2);
            sx[jj] = sx[jj+mark];

        __syncthreads();

        hcp::gpu::cuda::s4::LinearFit<double_t>(X, sx, sx_size, &mu, &beta);

        //std::cout << "y = " << mu_t << "x + " << beta_t << std::endl;
        //std::cout << "eValue: " << pow(10, hyp * mu_t + beta_t) * vaa << std::endl;
    }

    /* Estimate the log(s(x)) */
    double_t lgs_x = (mu * hyp) + beta;

    /* Compute the e(x) = n * s(x) = n * 10^(lg(s(x))) */
    evalues[bid] = cpsms[bid] * pow(10.0, lgs_x);

}

// -------------------------------------------------------------------------------------------- //

__host__ status_t processResults(Index *index, Queries<spectype_t> *gWorkPtr, int currspecID)
{
    status_t status = SLM_SUCCESS;

    // get driver object
    auto driver = hcp::gpu::cuda::driver::get_instance();

    // get scorecard instance
    auto d_Scores = hcp::gpu::cuda::s3::getScorecard();

    auto d_evalues = getd_eValues();

    int numSpecs = gWorkPtr->numSpecs;

    int blockSize = std::min(1024, HISTOGRAM_SIZE);
    // short min_cpsm = params.min_cpsm;

    // make sure the data stream is in sync
    driver->stream_sync(DATA_STREAM);

#if defined (TAILFIT) || true

    // use function pointers to point to the correct overload
    auto TailFit_ptr = static_cast<void (*)(dScores_t *, double *)>(&TailFit);

    // IMPORTANT: make sure at least 32KB+ shared memory is available to the TailFit kernel
    hipFuncSetAttribute(reinterpret_cast<const void*>(*TailFit_ptr), hipFuncAttributeMaxDynamicSharedMemorySize, KBYTES(48));

    // the tailfit kernel
    hcp::gpu::cuda::s4::TailFit<<<numSpecs, blockSize, KBYTES(48), driver->get_stream(SEARCH_STREAM)>>>(d_Scores, d_evalues);
#else
    // IMPORTANT: make sure at least 32KB+ shared memory is available to the logWeibullfit kernel
    //hipFuncSetAttribute(reinterpret_cast<const void*>(logWeibullFit), hipFuncAttributeMaxDynamicSharedMemorySize, KBYTES(48));

    // the logWeibullfit kernel
    //hcp::gpu::cuda::s4::logWeibullFit<<<numSpecs, blockSize, KBYTES(48), driver->get_stream(SEARCH_STREAM)>>>(d_Scores, d_evalues, min_cpsm);
#endif // TAILFIT

    // synchronize the search stream
    driver->stream_sync(SEARCH_STREAM);

    // host dScores
    hcp::gpu::cuda::s3::dScores *h_dScores = new hcp::gpu::cuda::s3::dScores();

    // copy pointers from the device
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::D2H(h_dScores, d_Scores, 1, driver->stream[DATA_STREAM]));

    // asynchronously copy the dhCell and cpsms to hostmem for writing to file
    dhCell *h_topscore = new dhCell[numSpecs];
    int *h_cpsms = new int[numSpecs];
    double *h_evalues = new double[numSpecs];

    // transfer data to host
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::D2H(h_topscore, h_dScores->topscore, numSpecs, driver->stream[DATA_STREAM]));
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::D2H(h_cpsms, h_dScores->cpsms, numSpecs, driver->stream[DATA_STREAM]));
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::D2H(h_evalues, d_evalues, numSpecs, driver->stream[DATA_STREAM]));

    // synchronize the stream
    driver->stream_sync(DATA_STREAM);

    // write all results to file
    for (int s = 0; s < numSpecs; s++)
    {
        if (h_evalues[s] < params.expect_max)
        {
            hCell psm;

            psm.idxoffset = h_topscore[s].idxoffset;
            psm.hyperscore = h_topscore[s].hyperscore;
            psm.sharedions = h_topscore[s].sharedions;
            psm.psid = h_topscore[s].psid;
            psm.totalions = (index[psm.idxoffset].pepIndex.peplen - 1) * params.maxz * iSERIES;
            psm.rtime = gWorkPtr->rtimes[s];
            psm.pchg = gWorkPtr->charges[s];
            psm.fileIndex = gWorkPtr->fileNum;

            /* Printing the scores in OpenMP mode */
            status = DFile_PrintScore(index, currspecID + s, gWorkPtr->precurse[s], &psm, h_evalues[s], h_cpsms[s]);
        }
    }

    // delete the temp memory
    delete[] h_topscore;
    delete[] h_cpsms;
    delete[] h_evalues;

    delete h_dScores;

    h_topscore = nullptr;
    h_cpsms = nullptr;
    h_evalues = nullptr;
    h_dScores = nullptr;

    return status;
}

// -------------------------------------------------------------------------------------------- //

#ifdef USE_MPI

__host__ status_t getIResults(Index *index, Queries<spectype_t> *gWorkPtr, int currSpecID, hCell *CandidatePSMS)
{
    status_t status = SLM_SUCCESS;

    ebuffer *liBuff = nullptr;
    partRes *txArray = nullptr;

    liBuff = new ebuffer;

    txArray = liBuff->packs;
    liBuff->isDone = false;
    liBuff->batchNum = gWorkPtr->batchNum;

    int numSpecs = gWorkPtr->numSpecs;

   // get driver object
    auto driver = hcp::gpu::cuda::driver::get_instance();

    // get device scorecard instance
    auto d_Scores = hcp::gpu::cuda::s3::getScorecard();

    // host dScores
    dScores_t *h_dScores = new hcp::gpu::cuda::s3::dScores();

    // copy pointers from the device
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::D2H(h_dScores, d_Scores, 1, driver->stream[DATA_STREAM]));

    driver->stream_sync(DATA_STREAM);

    // asynchronously copy the dhCell and cpsms to hostmem for writing to file
    dhCell *h_topscore = new dhCell[numSpecs];
    int *h_cpsms = new int[numSpecs];
    double *h_survival = new double [HISTOGRAM_SIZE * numSpecs];

    // transfer data to host
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::D2H(h_topscore, h_dScores->topscore, numSpecs, driver->stream[DATA_STREAM]));
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::D2H(h_cpsms, h_dScores->cpsms, numSpecs, driver->stream[DATA_STREAM]));
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::D2H(h_survival, h_dScores->survival, HISTOGRAM_SIZE * numSpecs, driver->stream[DATA_STREAM]));

    // synchronize the stream
    driver->stream_sync(DATA_STREAM);

    // process all results
    for (int s = 0; s < numSpecs; s++)
    {
        if (h_cpsms[s] >= 1)
        {
            hCell psm;

            psm.idxoffset = h_topscore[s].idxoffset;
            psm.hyperscore = h_topscore[s].hyperscore;
            psm.sharedions = h_topscore[s].sharedions;
            psm.psid = h_topscore[s].psid;
            psm.totalions = (index[psm.idxoffset].pepIndex.peplen - 1) * params.maxz * iSERIES;
            psm.rtime = gWorkPtr->rtimes[s];
            psm.pchg = gWorkPtr->charges[s];
            psm.fileIndex = gWorkPtr->fileNum;

            /* Put it in the list */
            CandidatePSMS[s] = psm;

            // FIXME: Is this needed?
            //resPtr->maxhypscore = (psm.hyperscore * 10 + 0.5);

            auto &&minnext = expeRT::StoreIResults(&h_survival[s * HISTOGRAM_SIZE], s, h_cpsms[s], liBuff);

            /* Fill in the Tx array cells */
            txArray[s].min  = minnext[0]; // minhypscore
            txArray[s].max2 = minnext[1]; // nexthypscore
            txArray[s].max  = psm.hyperscore;
            txArray[s].N    = h_cpsms[s];
            txArray[s].qID  = currSpecID + s;
        }
    }

    // add liBuff to sub-task K
    if (params.nodes > 1)
    {
        liBuff->currptr = numSpecs * Xsamples * sizeof(ushort_t);
        AddliBuff(liBuff);
    }

    // delete the temp memory
    delete[] h_topscore;
    delete[] h_cpsms;
    delete[] h_survival;

    delete h_dScores;

    h_topscore = nullptr;
    h_cpsms = nullptr;
    h_survival = nullptr;
    h_dScores = nullptr;

    return status;
}

#endif // USE_MPI

// -------------------------------------------------------------------------------------------- //

template <class T>
__device__ void LinearFit(T* x, T* y, int n, double *a, double *b)
{
    double bot;
    double top;
    double xbar;
    double ybar;

    //
    //  Special case.
    //
    if (n == 1)
    {
        *a = 0.0;
        *b = y[0];
    }
    else
    {
        //
        //  Average X and Y.
        //
        xbar = 0.0;
        ybar = 0.0;

        hcp::gpu::cuda::s4::XYbar<double>(x, y, n, xbar, ybar);

        xbar = xbar / (double) n;
        ybar = ybar / (double) n;

        //
        //  Compute Beta.
        //

        top = 0.0;
        bot = 0.0;

        hcp::gpu::cuda::s4::TopBot<double>(x, y, n, xbar, ybar, top, bot);

        *a = top / bot;
        *b = ybar - (*a) * xbar;
    }

    return;
}

// -------------------------------------------------------------------------------------------- //

__host__ void processResults(double *h_data, float *h_hyp, int *h_cpsms, double *h_evalues, int bsize)
{
    double_t *d_data;
    int *d_cpsms;
    float *d_hyp;
    double *d_evalues;

    // driver
    auto driver = hcp::gpu::cuda::driver::get_instance();

    // allocate device memory
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_allocate_async(d_data, HISTOGRAM_SIZE * bsize, driver->get_stream()));
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_allocate_async(d_cpsms, bsize, driver->get_stream()));
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_allocate_async(d_hyp, bsize, driver->get_stream()));
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_allocate_async(d_evalues, bsize, driver->get_stream()));

    // H2D
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::H2D(d_data, h_data, HISTOGRAM_SIZE * bsize, driver->get_stream()));
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::H2D(d_cpsms, h_cpsms, bsize, driver->get_stream()));
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::H2D(d_hyp, h_hyp, bsize, driver->get_stream()));

    int blockSize = std::min(1024, HISTOGRAM_SIZE);

#if defined (TAILFIT) || true

    // use function pointers to point to the correct overload
    auto TailFit_ptr = static_cast<void (*)(double *, float *, int *, double *)>(&TailFit);

    // IMPORTANT: make sure at least 32KB+ shared memory is available to the TailFit kernel
    hipFuncSetAttribute(reinterpret_cast<const void*>(*TailFit_ptr), hipFuncAttributeMaxDynamicSharedMemorySize, KBYTES(48));

    // the tailfit kernel
    hcp::gpu::cuda::s4::TailFit<<<bsize, blockSize, KBYTES(48), driver->get_stream()>>>(d_data, d_hyp, d_cpsms, d_evalues);
#else
    // IMPORTANT: make sure at least 32KB+ shared memory is available to the logWeibullfit kernel
    //hipFuncSetAttribute(reinterpret_cast<const void*>(logWeibullFit), hipFuncAttributeMaxDynamicSharedMemorySize, KBYTES(48));

    // the logWeibullfit kernel
    //hcp::gpu::cuda::s4::logWeibullFit<<<numSpecs, blockSize, KBYTES(48), driver->get_stream(SEARCH_STREAM)>>>(d_Scores, d_evalues, min_cpsm);
#endif // TAILFIT

    // D2H
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::D2H(h_evalues, d_evalues, bsize, driver->get_stream()));

    // free device memory
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_free_async(d_evalues, driver->get_stream()));
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_free_async(d_data, driver->get_stream()));
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_free_async(d_cpsms, driver->get_stream()));
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_free_async(d_hyp, driver->get_stream()));

    // synchronize
    driver->stream_sync();
}

// -------------------------------------------------------------------------------------------- //

} // namespace s4
} // namespace cuda
} // namespace gpu
} // namespace hcp
