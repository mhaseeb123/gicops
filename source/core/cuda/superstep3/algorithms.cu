#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2022 Muhammad Haseeb, and Fahad Saeed
 * Florida International University, Miami, FL
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <https://www.gnu.org/licenses/>.
 *
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "cuda/driver.hpp"
#include "cuda/superstep3/kernel.hpp"

namespace hcp
{

namespace gpu
{

namespace cuda
{

namespace s3
{

// -------------------------------------------------------------------------------------------- //

__device__ void lower_bound(uint_t *data, int size, int *lbound, int t)
{
    int n = size;
 
    // if there are no elements in nums
    if(n==0 || data[n-1] < t)
        *lbound = n;
    else if (data[0] >= t)
        *lbound = 0;
    else
    {
        // initialized low(l), and high(r)
        int l=0;
        int r = n-1;
        int m = l + (r-l)/2;

        while(l <= r)
        {
            m = l + (r-l)/2;
            if(data[m]>=t)
                r = m-1;
            else
                l = m+1;
        }

        *lbound = l;
    }

    return;
}

// -------------------------------------------------------------------------------------------- //

__device__ void upper_bound(uint_t *data, int size, int *ubound, int t)
{
    int n = size;

    // if there are no elements in nums
    if(n==0 || data[0] > t)
        *ubound = -1;
    else if (data[n-1] <= t)
        *ubound = n-1;
    else
    {
        // initialized low(l), and high(r)
        int l=0;
        int r = n-1;
        int m = l + (r-l)/2;

        while(l <= r)
        {
            m = l + (r-l)/2;
            if(data[m]<=t)
                l = m+1;
            else
                r = m-1;
        }

        *ubound = r;
    }

    return;
}

// -------------------------------------------------------------------------------------------- //

__device__ void compute_minmaxions(int *minions, int *maxions, int *QAPtr, uint *d_bA, uint *d_iA, int dF, int qspeclen, int speclen, int minlimit, int maxlimit, int maxmass, int scale)
{
    int tid = threadIdx.x;
    short bucket = (2*dF+1);

    // total ions with mass +-dF
    int irange = qspeclen * bucket;

    for (int a = tid; a < irange; a+=blockDim.x)
    {
        minions[a] = 0;
        maxions[a] = -1;
    }

    __syncthreads();

    // for all ions
    for (int ion = tid; ion < irange ; ion += blockDim.x)
    {
        // main ion
        auto myion = QAPtr[ion/bucket];
        // dF offset
        int myion_offset = (ion % bucket) - dF;

        // new ion mass
        auto qion = myion + myion_offset;

        int maxionmass = (maxmass * scale) - 1 - dF;

        //printf("tid: %d, qion: %d\n", tid, qion);

        // check for legal ion mass
        if (myion > dF && myion <= maxionmass)
        {
            // locate iAPtr start and end
            uint_t *data_ptr = d_iA + d_bA[qion];
            int data_size = d_bA[qion+1] - d_bA[qion];

            // if no ions in the bin
            if (data_size < 1)
                continue;

            // lowerbound limit
            int target = minlimit * speclen;

            // compute lower bound
            lower_bound(data_ptr, data_size, &minions[ion], target);

            __threadfence_block();

            // upperbound limit
            target = (((maxlimit + 1) * speclen) - 1);

            upper_bound(data_ptr, data_size, &maxions[ion], target);

            __threadfence_block();
        }
    }

    __syncthreads();

    return;

}

// -------------------------------------------------------------------------------------------- //

__device__ void getMaxdhCell(dhCell &topscores, dhCell &out)
{
    int tid = threadIdx.x;
    int warpsize = 32;
    int warpId = tid / warpsize;
    int laneId = tid % warpsize;
    int nwarps = blockDim.x / warpsize;
    int nthreads = blockDim.x;

    // get the max element
    int myIdx = tid;
    float myhScore = topscores.hyperscore;

    unsigned mask  = __ballot_sync(0xffffffff, tid < nthreads);

    for(int offset = warpsize / 2; offset > 0; offset /= 2)
    {
        float tempScore = __shfl_down_sync(mask, myhScore, offset);
        int tempIdx = __shfl_down_sync(mask, myIdx, offset);

        if (tempScore > myhScore)
        {
            myhScore = tempScore;
            myIdx = tempIdx;
        }
    }

    __syncthreads();

    __shared__ float lochScore[32];
    __shared__ int locIdx[32];
    __shared__ dhCell thetopscore;

    if (laneId == 0)
    {
        lochScore[warpId] = myhScore;
        locIdx[warpId] = myIdx;
    }

    __syncthreads();

    if (tid < nwarps)
    {
        myhScore = lochScore[tid];
        myIdx = locIdx[tid];
    }
    else
    {
        myhScore = 0;
        myIdx = -1;
    }

    __syncthreads();

    if (warpId == 0)
    {
        unsigned int mask  = __ballot_sync(0xffffffff, tid < nwarps);

        for(int offset = warpsize / 2; offset > 0; offset /= 2)
        {
            float tempScore = __shfl_down_sync(mask, myhScore, offset);
            int tempIdx = __shfl_down_sync(mask, myIdx, offset);

            if (tempScore > myhScore)
            {
                myhScore = tempScore;
                myIdx = tempIdx;
            }
        }
    }

    __syncthreads();

    // the final value should be at location zero
    if (tid == 0)
        locIdx[0] = myIdx;

    __syncthreads();

    // write the topscore at the shared memory
    if (tid == locIdx[0])
    {
        thetopscore.hyperscore = topscores.hyperscore;
        thetopscore.psid = topscores.psid;
        thetopscore.idxoffset = topscores.idxoffset;
        thetopscore.sharedions = topscores.sharedions;
    }

    __syncthreads();

    // pick the topscore from the shared memory
    out.hyperscore = thetopscore.hyperscore;
    out.psid = thetopscore.psid;
    out.idxoffset = thetopscore.idxoffset;
    out.sharedions = thetopscore.sharedions;

    return;
}

// -------------------------------------------------------------------------------------------- //

template <typename T>
__device__ void blockSum(T val, T &sum)
{
    short tid = threadIdx.x;
    short warpsize = 32;
    short warpId = tid / warpsize;
    short laneId = tid % warpsize;
    short bsize = blockDim.x;
    short nwarps = bsize / warpsize;
    nwarps += (bsize % warpsize) ? 1 : 0;

    // sum a warp
    unsigned mask  = __ballot_sync(0xffffffff, tid < bsize);

    for (int offset = warpsize / 2; offset > 0; offset /= 2)
    {
        T tempVal = __shfl_down_sync(mask, val, offset);
        val += tempVal;
    }

    __syncthreads();

    // sum a block
    __shared__ T bSum[32];

    if (laneId == 0)
        bSum[warpId] = val;

    __syncthreads();

    if (tid < nwarps)
        val = bSum[tid];
    else
        val = 0;

    if (warpId == 0)
    {
        mask  = __ballot_sync(0xffffffff, tid < nwarps);

        for (int offset = warpsize / 2; offset > 0; offset /= 2)
        {
            T tempVal = __shfl_down_sync(mask, val, offset);
            val += tempVal;
        }
    }

    if (tid == 0)
        bSum[0] = val;

    __syncthreads();

    // fetch the final sum from the shared memory
    sum = bSum[0];

    return;
}

// -------------------------------------------------------------------------------------------- //

// instantiate the templates
template __device__ void blockSum<int>(int val, int &sum);

// -------------------------------------------------------------------------------------------- //

} // namespace s3

} // namespace cuda

} // namespace gpu

} // namespace hcp