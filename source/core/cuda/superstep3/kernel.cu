/*
 * Copyright (C) 2022 Muhammad Haseeb, and Fahad Saeed
 * Florida International University, Miami, FL
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <https://www.gnu.org/licenses/>.
 *
 */

#include <hip/hip_runtime.h>
#include <string>
#include <thrust/fill.h>
#include <thrust/gather.h>
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/sequence.h>
#include <thrust/distance.h>
#include <thrust/binary_search.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>

#include <iostream>
#include <thread>

#include "cuda/driver.hpp"
#include "cuda/superstep3/kernel.hpp"

using namespace std;

extern gParams params;

// -------------------------------------------------------------------------------------------- //

namespace hcp 
{

namespace gpu
{

namespace cuda
{

namespace s3
{

// -------------------------------------------------------------------------------------------- //


} // namespace s3

} // namespace cuda

} // namespace gpu

} // namespace hcp