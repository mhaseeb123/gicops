#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2022 Muhammad Haseeb, and Fahad Saeed
 * Florida International University, Miami, FL
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License, or
 * (at your option) any later version.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <https://www.gnu.org/licenses/>.
 *
 */

#include <hip/hip_runtime.h>
#include <string>
#include <thrust/fill.h>
#include <thrust/gather.h>
#include <thrust/sort.h>
#include <thrust/scan.h>
#include <thrust/sequence.h>
#include <thrust/distance.h>
#include <thrust/binary_search.h>
#include <thrust/functional.h>
#include <thrust/execution_policy.h>

#include <iostream>
#include <thread>

#include "cuda/driver.hpp"

#include "cuda/superstep1/kernel.hpp"
#include "cuda/superstep3/kernel.hpp"
#include "cuda/superstep4/kernel.hpp"

using namespace std;

// -------------------------------------------------------------------------------------------- //

// include the CUDA constant memory objects for A generation
// array to store PTM masses
extern __constant__ float_t modMass[ALPHABETS];

// amino acid masses
extern __constant__ float_t aaMass[ALPHABETS];

// static mod masses
extern __constant__ float_t statMass[ALPHABETS];

// log(factorial(n))
__constant__ double_t d_lgFact[hcp::utils::maxshp];


// -------------------------------------------------------------------------------------------- //

// host side global parameters
extern gParams params;

// -------------------------------------------------------------------------------------------- //

namespace hcp 
{

namespace gpu
{

namespace cuda
{

// -------------------------------------------------------------------------------------------- //

namespace s1
{

extern __device__ int log2ceil(unsigned long long x);

}

// -------------------------------------------------------------------------------------------- //

namespace s3
{

//
// CUDA kernel declarations
//

// -------------------------------------------------------------------------------------------- //

__global__ void SpSpGEMM(spectype_t *dQ_moz, spectype_t *dQ_intensity, uint_t *dQ_idx, int *dQ_minlimits, int *dQ_maxlimits, 
                        uint_t* d_bA, uint_t *d_iA, int iter, BYC *bycP, int maxchunk, double *d_survival, int *d_cpsms, 
                        dhCell *d_topscore, int dF, int speclen, int maxmass, int scale, short min_shp, int ixx);

// database search kernel host wrapper
__host__ status_t SearchKernel(Queries<spectype_t> *, int, int);

// compute min and max limits for the spectra
__host__ status_t MinMaxLimits(Queries<spectype_t> *, Index *, double dM);

template <typename T>
__global__ void vector_plus_constant(T *vect, T val, int size);


__global__ void resetdScores(double *survival, int *cpsms, dhCell *topscores);

extern __device__ void compute_minmaxions(int *minions, int *maxions, int *QAPtr, uint *d_bA, uint *d_iA, int dF, int qspeclen, int speclen, int minlimit, int maxlimit, int maxmass, int scale);

extern __device__ void getMaxdhCell(dhCell *topscores, dhCell *out);

extern __device__ void getMinsurvival(double_t *survival, double_t *out);

template <typename T>
extern __device__ void blockSum(T val, T &sum);


// -------------------------------------------------------------------------------------------- //

dScores::dScores()
{
    auto driver = hcp::gpu::cuda::driver::get_instance();

    // allocate memory for the scores
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_allocate_async(this->survival, HISTOGRAM_SIZE * QCHUNK, driver->stream[DATA_STREAM]));
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_allocate_async(this->topscore, QCHUNK, driver->stream[DATA_STREAM]));
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_allocate_async(this->cpsms, QCHUNK, driver->stream[DATA_STREAM]));

    hcp::gpu::cuda::s3::resetdScores<<<256, 256, KBYTES(1), driver->stream[DATA_STREAM]>>>(this->survival, this->cpsms, this->topscore);

    driver->stream_sync(DATA_STREAM);
}

// -------------------------------------------------------------------------------------------- //

dScores::~dScores()
{
    auto driver = hcp::gpu::cuda::driver::get_instance();

    // free all memory
    if (this->survival)
    {
        hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_free_async(this->survival, driver->stream[DATA_STREAM]));
        this->survival = nullptr;
    }

    if (this->topscore)
    {
        hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_free_async(this->topscore, driver->stream[DATA_STREAM]));
        this->topscore = nullptr;
    }

    if (this->cpsms)
    {
        hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_free_async(this->cpsms, driver->stream[DATA_STREAM]));
        this->cpsms = nullptr;
    }

    driver->stream_sync(DATA_STREAM);
}

// -------------------------------------------------------------------------------------------- //

template <typename T>
dQueries<T>::dQueries()
{
    auto driver = hcp::gpu::cuda::driver::get_instance();

    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_allocate_async(this->idx, QCHUNK+1, driver->stream[DATA_STREAM]));
    //hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_allocate_async(this->precurse, QCHUNK, driver->stream[DATA_STREAM]));
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_allocate_async(this->minlimits, QCHUNK, driver->stream[DATA_STREAM]));
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_allocate_async(this->maxlimits, QCHUNK, driver->stream[DATA_STREAM]));
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_allocate_async(this->moz, QCHUNK * QALEN, driver->stream[DATA_STREAM]));
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_allocate_async(this->intensity, QCHUNK * QALEN, driver->stream[DATA_STREAM]));

    numPeaks        = 0;
    numSpecs        = 0;
}

// -------------------------------------------------------------------------------------------- //

template <typename T>
void dQueries<T>::H2D(Queries<T> *rhs)
{
    auto driver = hcp::gpu::cuda::driver::get_instance();
    int chunksize = rhs->numSpecs;
    
    this->numSpecs = rhs->numSpecs;
    this->numPeaks = rhs->numPeaks;

    hcp::gpu::cuda::error_check(hcp::gpu::cuda::H2D(this->moz, rhs->moz, this->numPeaks, driver->stream[DATA_STREAM]));
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::H2D(this->intensity, rhs->intensity, this->numPeaks, driver->stream[DATA_STREAM]));
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::H2D(this->idx, rhs->idx, chunksize+1, driver->stream[DATA_STREAM]));
    //hcp::gpu::cuda::error_check(hcp::gpu::cuda::H2D(this->precurse, rhs.precurse, chunksize, driver->stream[DATA_STREAM]));

    // driver->stream_sync(DATA_STREAM);
}

// -------------------------------------------------------------------------------------------- //

template <typename T>
dQueries<T>::~dQueries()
{
    auto driver = hcp::gpu::cuda::driver::get_instance();

    numPeaks = 0;
    numSpecs = 0;

    /* Deallocate the memory */
    if (this->moz != nullptr)
    {
        hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_free_async(this->moz, driver->stream[DATA_STREAM]));
        this->moz = nullptr;
    }
    if (this->intensity != nullptr)
    {
        hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_free_async(this->intensity, driver->stream[DATA_STREAM]));
        this->intensity = nullptr;
    }
    if (this->minlimits != nullptr)
    {
        hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_free_async(this->minlimits, driver->stream[DATA_STREAM]));
        this->minlimits = nullptr;
    }
    if (this->maxlimits != nullptr)
    {
        hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_free_async(this->maxlimits, driver->stream[DATA_STREAM]));
        this->maxlimits = nullptr;
    }

    if (this->idx != nullptr)
    {
        hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_free_async(this->idx, driver->stream[DATA_STREAM]));
        this->idx = nullptr;
    }

    driver->stream_sync(DATA_STREAM);
}

// -------------------------------------------------------------------------------------------- //

__host__ status_t initialize()
{
    // static instance of the log(factorial(x)) array
    static auto h_lgfact = hcp::utils::lgfact<hcp::utils::maxshp>();

    // copy to CUDA constant arrays
    hcp::gpu::cuda::error_check(hipMemcpyToSymbol(HIP_SYMBOL(d_lgFact), &h_lgfact.val, sizeof(double_t) * hcp::utils::maxshp)); 

    return SLM_SUCCESS;

}

// -------------------------------------------------------------------------------------------- //

__global__ void reset_BYC(BYC *data, int N)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = tid; i < N; i += stride)
    {
        data[i].bc = 0;
        data[i].yc = 0;
        data[i].ibc = 0;
        data[i].iyc = 0;
    }
}

// -------------------------------------------------------------------------------------------- //

std::pair<BYC *, int>& getBYC(int chunksize)
{
    auto driver = hcp::gpu::cuda::driver::get_instance();

    static std::pair<BYC *, int> bycPair;
    static BYC *d_BYC = nullptr;
    static int maxchunk = 0;

    if (!d_BYC)
    {
        maxchunk = chunksize;

        if (!maxchunk)
            std::cout << "Error: getBYC: chunksize is zero" << std::endl;
        else
        {
            hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_allocate_async<BYC>(d_BYC, maxchunk * SEARCHINSTANCES, driver->stream[DATA_STREAM]));

            int nblocks = (maxchunk * SEARCHINSTANCES + 1023) / 1024;
            int nthreads = 1024;

            reset_BYC<<<nblocks, nthreads, KBYTES(1), driver->stream[DATA_STREAM]>>>(d_BYC, maxchunk * SEARCHINSTANCES);
        }
    }

    driver->stream_sync(DATA_STREAM);

    // update the pair
    bycPair = make_pair(d_BYC, maxchunk);

    // return the static pair
    return bycPair;
}

// -------------------------------------------------------------------------------------------- //

void freeBYC()
{
    auto driver = hcp::gpu::cuda::driver::get_instance();
    
    auto pBYC = getBYC();

    auto d_BYC = std::get<0>(pBYC);
    auto maxchunk = std::get<1>(pBYC);

    if (d_BYC)
        // free the d_Scores
        hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_free_async(d_BYC, driver->stream[DATA_STREAM]));

    if (maxchunk)
        maxchunk = 0;

    driver->stream_sync(DATA_STREAM);

    // set to nullptr
    d_BYC = nullptr;
}

// -------------------------------------------------------------------------------------------- //

dScores *& getScorecard()
{
    auto driver = hcp::gpu::cuda::driver::get_instance();
    static dScores *d_Scores = nullptr;

    if (!d_Scores)
        d_Scores = new dScores();

    return d_Scores;
}

// -------------------------------------------------------------------------------------------- //

void freeScorecard()
{
    auto driver = hcp::gpu::cuda::driver::get_instance();
    
    auto&& d_Scores = getScorecard();

    if (d_Scores)
    {
        delete d_Scores;
        d_Scores = nullptr;
    }
}

// -------------------------------------------------------------------------------------------- //

__host__ dQueries<spectype_t> *& getdQueries()
{
    static thread_local dQueries<spectype_t> *dqueries = nullptr;
    
    if (!dqueries)
        dqueries = new dQueries<spectype_t>();

    return dqueries;
}

// -------------------------------------------------------------------------------------------- //

__host__ void freedQueries()
{
    auto &&dqueries = getdQueries();

    if (dqueries)
    {
        delete dqueries;
        dqueries = nullptr;
    }
}

// -------------------------------------------------------------------------------------------- //

// the database search kernel
__host__ status_t search(Queries<spectype_t> *gWorkPtr, Index *index, uint_t idxchunks, int gpucurrSpecID, hCell *CandidatePSMS)
{
    status_t status = SLM_SUCCESS;

    auto dqueries = getdQueries();

    // transfer experimental data to device
    dqueries->H2D(gWorkPtr);

    // number of spectra in the current batch
    //int nspectra = gWorkPtr->numSpecs;

    // get instance of the driver
    static thread_local auto driver = hcp::gpu::cuda::driver::get_instance();

    // sync all data streams
    driver->stream_sync(DATA_STREAM);

    // search for each database chunk (by length)
    for (int i = 0; i < idxchunks ; i++)
    {
        // get the current index portion (by length)
        Index *curr_index = &index[i];

        // FIXME: make min-max limits for the spectra in the current chunk
        status = hcp::gpu::cuda::s3::MinMaxLimits(gWorkPtr, curr_index, params.dM);

        uint_t speclen = (curr_index->pepIndex.peplen - 1) * params.maxz * iSERIES;

        // construct for each intra-chunk (within each length)
        for (int chno = 0; chno < index->nChunks && status == SLM_SUCCESS; chno++)
        {
    
#if 1       // TODO: Leave this or remove this

            // build the AT columns. i.e. the iAPtr
            status = hcp::gpu::cuda::s1::ConstructIndexChunk(curr_index, chno, true);
            auto d_iA = hcp::gpu::cuda::s1::getATcols();

#else

            /* Check if this chunk is the last chunk */
            uint_t nsize = ((chno == curr_index->nChunks - 1) && (curr_index->nChunks > 1))?
                   curr_index->lastchunksize : curr_index->chunksize;

            uint_t *iAPtr = curr_index->ionIndex[chno].iA;
            uint_t iAsize = nsize * speclen;

            // copy the At columns to the device instead
            auto d_iA = hcp::gpu::cuda::s1::getATcols(iAsize);

            hcp::gpu::cuda::error_check(hcp::gpu::cuda::H2D(d_iA, iAPtr, iAsize, driver->stream[DATA_STREAM]));
#endif // 1

            // copy the At rows to device 
            auto d_bA = hcp::gpu::cuda::s1::getbA();
            uint_t bAsize = ((uint_t)(params.max_mass * params.scale)) + 1;

            hcp::gpu::cuda::error_check(hcp::gpu::cuda::H2D(d_bA, curr_index->ionIndex[chno].bA, bAsize, driver->stream[DATA_STREAM]));

            // synch both streams
            driver->stream_sync(DATA_STREAM);

            // search against the database
            status = hcp::gpu::cuda::s3::SearchKernel(gWorkPtr, speclen, i);

            // free the AT columns
            hcp::gpu::cuda::s1::freeATcols();
        }
    }

#ifdef USE_MPI

    if (params.nodes > 1)
        status = hcp::gpu::cuda::s4::getIResults(index, gWorkPtr, gpucurrSpecID, CandidatePSMS);
    else
#else
        // combine the results
        status = hcp::gpu::cuda::s4::processResults(index, gWorkPtr, gpucurrSpecID);
#endif // USE_MPI

    hcp::gpu::cuda::s3::reset_dScores();

    return status;
}

// -------------------------------------------------------------------------------------------- //

__host__ status_t MinMaxLimits(Queries<spectype_t> *h_WorkPtr, Index *index, double dM)
{
    status_t status = SLM_SUCCESS;
    auto driver = hcp::gpu::cuda::driver::get_instance();

    auto d_WorkPtr = getdQueries();

    // extract all peptide masses in an array to simplify computations
    float_t *h_mzs; // = new float_t[index->lcltotCnt];
    hcp::gpu::cuda::host_pinned_allocate<float_t>(h_mzs, index->lcltotCnt);

    // simplify the peptide masses
    for (int i = 0; i < index->lcltotCnt; i++)
        h_mzs[i] = index->pepEntries[i].Mass;

    auto size = index->lcltotCnt;

    // initialize device vector with mzs
    float *d_mzs = nullptr;
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_allocate_async(d_mzs, size, driver->stream[SEARCH_STREAM]));
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::H2D(d_mzs, h_mzs, size, driver->stream[SEARCH_STREAM]));

    float *d_precurse = nullptr;
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_allocate_async(d_precurse, h_WorkPtr->numSpecs, driver->stream[SEARCH_STREAM]));
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::H2D(d_precurse, h_WorkPtr->precurse, h_WorkPtr->numSpecs, driver->stream[SEARCH_STREAM]));

    const int nthreads = 1024;
    int nblocks = h_WorkPtr->numSpecs / 1024;
    nblocks += (h_WorkPtr->numSpecs % 1024 == 0)? 0 : 1;

    // add -dM to set for minlimit
    hcp::gpu::cuda::s3::vector_plus_constant<<<nblocks, nthreads, KBYTES(1), driver->get_stream(SEARCH_STREAM)>>>(d_precurse, (float)(-dM), h_WorkPtr->numSpecs);

    // binary search the start of each ion and store in minlimits
    thrust::lower_bound(thrust::device.on(driver->get_stream(SEARCH_STREAM)), d_mzs, d_mzs + size, d_precurse, d_precurse + h_WorkPtr->numSpecs, d_WorkPtr->minlimits);

    // add -dM to set for minlimit
    hcp::gpu::cuda::s3::vector_plus_constant<<<nblocks, nthreads, KBYTES(1), driver->get_stream(SEARCH_STREAM)>>>(d_precurse, (float)(2*dM), h_WorkPtr->numSpecs);

    // binary search the end of each spectrum and store in maxlimits
    thrust::upper_bound(thrust::device.on(driver->get_stream(SEARCH_STREAM)), d_mzs, d_mzs + size, d_precurse, d_precurse + h_WorkPtr->numSpecs, d_WorkPtr->maxlimits);

    // d_mzs is no longer needed - free
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_free_async(d_mzs, driver->stream[SEARCH_STREAM]));
    hcp::gpu::cuda::error_check(hcp::gpu::cuda::device_free_async(d_precurse, driver->stream[SEARCH_STREAM]));

    // free the mzs array
    hcp::gpu::cuda::host_pinned_free(h_mzs);

    return status;
}

// -------------------------------------------------------------------------------------------- //

__host__ status_t SearchKernel(Queries<spectype_t> *gWorkPtr, int speclen, int ixx)
{
    status_t status = SLM_SUCCESS;

    auto d_WorkPtr = getdQueries();

    // number of spectra in the current batch
    int nspectra = gWorkPtr->numSpecs;

    auto d_iA = hcp::gpu::cuda::s1::getATcols();
    auto d_bA = hcp::gpu::cuda::s1::getbA();

    // get driver object
    auto driver = hcp::gpu::cuda::driver::get_instance();


    const int itersize = SEARCHINSTANCES;

    int niters = nspectra / itersize;
    niters += (nspectra % itersize == 0)? 0 : 1;

    // get resPtr instance
    auto d_Scores = hcp::gpu::cuda::s3::getScorecard();

    // get BYC scorecard
    auto pBYC = hcp::gpu::cuda::s3::getBYC();

    auto d_BYC = std::get<0>(pBYC);
    auto maxchunk = std::get<1>(pBYC);

    // set the shared memory to 48KB
    hipFuncSetAttribute(reinterpret_cast<const void*>(SpSpGEMM), hipFuncAttributeMaxDynamicSharedMemorySize, KBYTES(48));

    for (int iter = 0 ; iter < niters ; iter++)
    {
        int nblocks = itersize;
        int blocksize = 1024;

        // if last iteration, adjust the number of blocks
        if (iter == niters - 1)
            nblocks = nspectra - iter * itersize;

        hcp::gpu::cuda::s3::SpSpGEMM<<<nblocks, blocksize, KBYTES(48), driver->stream[SEARCH_STREAM]>>>(d_WorkPtr->moz, d_WorkPtr->intensity, d_WorkPtr->idx, d_WorkPtr->minlimits, d_WorkPtr->maxlimits, d_bA, d_iA, iter * itersize, d_BYC, maxchunk, d_Scores->survival, d_Scores->cpsms, d_Scores->topscore, params.dF, speclen, params.max_mass, params.scale, params.min_shp, ixx);
    }

    // synchronize the stream
    driver->stream_sync(SEARCH_STREAM);

    return status;
}

// -------------------------------------------------------------------------------------------- //

__host__ void reset_dScores()
{
    auto d_Scores = hcp::gpu::cuda::s3::getScorecard();

    // get driver object
    auto driver = hcp::gpu::cuda::driver::get_instance();

    // reset the scorecard
    hcp::gpu::cuda::s3::resetdScores<<<256, 256, KBYTES(1), driver->stream[SEARCH_STREAM]>>>(d_Scores->survival, d_Scores->cpsms, d_Scores->topscore);

    driver->stream_sync(SEARCH_STREAM);
}

// -------------------------------------------------------------------------------------------- //

__global__ void resetdScores(double *survival, int *cpsms, dhCell *topscore)
{
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (int ik = tid; ik < QCHUNK * HISTOGRAM_SIZE; ik+=stride)
        survival[ik] = 0;

    for (int ik = tid; ik < QCHUNK; ik+=stride)
    {
        cpsms[ik] = 0;
        topscore[ik].hyperscore = 0;
        topscore[ik].psid = 0;
        topscore[ik].idxoffset = 0;
        topscore[ik].sharedions = 0;
    }
}

// -------------------------------------------------------------------------------------------- //

__global__ void SpSpGEMM(spectype_t *dQ_moz, spectype_t *dQ_intensity, uint_t *dQ_idx, int *dQ_minlimits,
                         int *dQ_maxlimits, uint_t* d_bA, uint_t *d_iA, int iter, BYC *bycP, 
                         int maxchunk, double *d_survival, int *d_cpsms, dhCell *d_topscore, int dF, 
                         int speclen, int maxmass, int scale, short min_shp, int ixx)
{
    BYC *bycPtr = &bycP[blockIdx.x * maxchunk];

    // get spectrum data
    int qnum = iter + blockIdx.x;

    auto *survival = &d_survival[qnum * HISTOGRAM_SIZE];
    auto *cpsms = d_cpsms + qnum;

    auto *QAPtr = dQ_moz + dQ_idx[qnum];
    auto *iPtr = dQ_intensity + dQ_idx[qnum];
    int qspeclen = dQ_idx[qnum + 1] - dQ_idx[qnum];
    int halfspeclen = speclen / 2;

    int minlimit = dQ_minlimits[qnum];
    int maxlimit = dQ_maxlimits[qnum] - 1; // maxlimit = upper_bound - 1

    // if maxlimit < minlimit then no point, just return
    if (maxlimit < minlimit)
        return;

    // shared memory
    extern __shared__ int shmem[];

    // size for minions and maxions
    const int minmaxsize = ((2 * dF) + 1) * QALEN;

    int *minions = &shmem[0];
    int *maxions = &minions[minmaxsize];

    // keys = ppid, vals = BYC for reduction
    int *keys    = &maxions[minmaxsize];
    BYC *vals    = (BYC*)&keys[blockDim.x];

    // setup shared memory here
    __syncthreads();

    // compute min and maxlimits for ions in minions and maxions
    hcp::gpu::cuda::s3::compute_minmaxions(minions, maxions, QAPtr, d_bA, d_iA, dF, qspeclen, speclen, minlimit, maxlimit, maxmass, scale);

    // iterate over all ions
    for (int k = 0; k < qspeclen; k++)
    {
        uint_t intn = iPtr[k];
        auto qion = QAPtr[k];

        // ion +-dF
        for (auto bin = qion - dF; bin < qion + 1 + dF; bin++)
        {
            short binidx = k *(2*dF + 1) + dF - (qion - bin);

            int off1 = minions[binidx];
            int off2 = maxions[binidx];

            int stt = d_bA[bin] + off1;
            int ends = d_bA[bin] + off2;

            int nions = ends - stt + 1;
            int ioniters = nions / blockDim.x;
            ioniters += (nions % blockDim.x) ? 1 : 0;

            int itnum = 0;

            //
            // fragment ion search loop
            //
            for (int ion = stt + threadIdx.x; itnum < ioniters; ion+= blockDim.x, itnum++)
            {
                int myKey = 0;
                BYC *myVal = nullptr;

                if (ion <= ends)
                {
                    uint_t raw = d_iA[ion];

                    /* Calculate parent peptide ID */
                    int_t ppid = (raw / speclen);

                    /* Calculate the residue */
                    int_t residue = (raw % speclen);

                    /* Either 0 or 1 */
                    int isY = residue / halfspeclen;
                    int isB = 1 - isY;

                    // key - ppid
                    myKey = ppid;

                    // write to keys
                    keys[threadIdx.x] = myKey;

                    /* Get the map element */
                    myVal = &vals[threadIdx.x];
                    myVal->bc = isB;
                    myVal->ibc = intn * isB;
                    myVal->yc = isY;
                    myVal->iyc = intn * isY;
                }

                __syncthreads();

                //
                // reduce the BYC elements to avoid 
                // race conditions and locking
                //

                // number of active threads
                int activethds = min(blockDim.x, ends - ion + threadIdx.x + 1);

                int iters = hcp::gpu::cuda::s1::log2ceil(activethds);

                // is this thread a part of a localized group (and not group leader)
                bool isGroup = false;

                // threadIdx.x is always the leader
                if (threadIdx.x > 0)
                    isGroup = (myKey == keys[threadIdx.x - 1]) || (threadIdx.x >= activethds);

                // the reduction loop by all threads
                for (int ij = 0; ij < iters; ij++)
                {
                    int offset = 1 << ij;

                    int newKey = 0;
                    BYC newVal;

                    // exchange values
                    if (threadIdx.x < (activethds - offset))
                    {
                        int idx = threadIdx.x + offset;
                        newKey = keys[idx];

                        // if the keys match, get the new value
                        if (newKey == myKey)
                        {
                            newVal.bc = vals[idx].bc;
                            newVal.ibc = vals[idx].ibc;
                            newVal.yc = vals[idx].yc;
                            newVal.iyc = vals[idx].iyc;
                        }
                    }

                    __syncthreads();

                    // write the sum to the shm
                    if (threadIdx.x < (activethds - offset))
                    {
                        if (newKey == myKey)
                        {
                            myVal->bc += newVal.bc;
                            myVal->ibc += newVal.ibc;
                            myVal->yc += newVal.yc;
                            myVal->iyc += newVal.iyc;
                        }
                    }

                    // sync threads
                    __syncthreads();
                }

                // only write to global memory if no group or the leader
                if (!isGroup)
                {
                    BYC *glob = &bycPtr[myKey];
                    glob->bc += myVal->bc;
                    glob->ibc += myVal->ibc;
                    glob->yc += myVal->yc;
                    glob->iyc += myVal->iyc;
                }
            }

            // synchronize
            __syncthreads();
        }
    }

    // reuse the shared memory
    int *histogram = &shmem[0];
    dhCell *topscores = (dhCell *)&histogram[HISTOGRAM_SIZE];

    // initialize
    for (int ij = threadIdx.x; ij < HISTOGRAM_SIZE; ij+=blockDim.x)
        histogram[ij] = 0;

    for (int ij = threadIdx.x; ij < blockDim.x; ij+=blockDim.x)
    {
        topscores[ij].hyperscore = 0;
        topscores[ij].psid = 0;
        topscores[ij].idxoffset = 0;
        topscores[ij].sharedions = 0;
    }

    // wait for shmem to be initialized
    __syncthreads();

    // thread local variable to store ncpsms
    int cpss = 0;

    /* Look for candidate PSMs */
    for (int_t it = minlimit + threadIdx.x; it <= maxlimit; it+= blockDim.x)
    {
        ushort_t bcc = bycPtr[it].bc;
        ushort_t ycc = bycPtr[it].yc;
        ushort_t shpk = bcc + ycc;

        // filter by the min shared peaks
        if (shpk >= min_shp) 
        {
            // Create a heap cell
            dhCell cell;

            // get the precomputed log(factorial(x))
            double_t h1 = d_lgFact[bcc] + d_lgFact[ycc];

            // Fill in the information
            cell.hyperscore = h1 + log10f(1 + bycPtr[it].ibc) + log10f(1 + bycPtr[it].iyc) - 6;

            // hyperscore < 0 means either b- or y- ions were not matched
            if (cell.hyperscore > 0)
            {
                cell.idxoffset = ixx;
                cell.psid = it;
                cell.sharedions = shpk;

                // increment local candidate psms by +1
                cpss +=1;

                // Update the histogram
                atomicAdd(&histogram[(int)(cell.hyperscore * 10 + 0.5)], 1);

                if (cell.hyperscore > topscores[threadIdx.x].hyperscore)
                {
                    topscores[threadIdx.x].hyperscore = cell.hyperscore;
                    topscores[threadIdx.x].psid       = cell.psid;
                    topscores[threadIdx.x].idxoffset  = cell.idxoffset;
                    topscores[threadIdx.x].sharedions = cell.sharedions;
                }
            }
        }
    }

    __syncthreads();

    dhCell l_topscore;

    // get max dhcell
    hcp::gpu::cuda::s3::getMaxdhCell(topscores, &l_topscore);

    __syncthreads();

    if (!threadIdx.x && (l_topscore.hyperscore > d_topscore[qnum].hyperscore))
    {
        d_topscore[qnum].hyperscore = l_topscore.hyperscore;
        d_topscore[qnum].psid       = l_topscore.psid;
        d_topscore[qnum].idxoffset  = l_topscore.idxoffset;
        d_topscore[qnum].sharedions = l_topscore.sharedions;
    }

    // local candidate psms
    int cpsms_loc = 0;

    // sum the local cpsms to get the count
    hcp::gpu::cuda::s3::blockSum(cpss, cpsms_loc);

    // write to the global memory
    if (!threadIdx.x)
        *cpsms = *cpsms + cpsms_loc;

    __syncthreads();

    // copy histogram to the global memory
    for (int ii = threadIdx.x; ii < HISTOGRAM_SIZE; ii+=blockDim.x)
        survival[ii] += histogram[ii];

    // reset the bycPtr
    for (int f = minlimit + threadIdx.x; f <= maxlimit; f += blockDim.x)
    {
        bycPtr[f].bc = 0;
        bycPtr[f].yc = 0;
        bycPtr[f].ibc = 0;
        bycPtr[f].iyc = 0;
    }

    return;
}

// -------------------------------------------------------------------------------------------- //

template <typename T>
__global__ void vector_plus_constant(T *vect, T val, int size)
{
    for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < size; i += blockDim.x * gridDim.x)
        vect[i] += val;
}

// -------------------------------------------------------------------------------------------- //

status_t deinitialize()
{
    hcp::gpu::cuda::s1::freeATcols();
    hcp::gpu::cuda::s1::freeFragIon();
    hcp::gpu::cuda::s1::freebA();
    // FIXME: why cuda error here
    // even if reallocate, then only one unit mem

    freedQueries();

    hcp::gpu::cuda::s4::freed_eValues();

    // sync all streams
    hcp::gpu::cuda::driver::get_instance()->all_streams_sync();

    return SLM_SUCCESS;
}

// -------------------------------------------------------------------------------------------- //

} // namespace s3

} // namespace cuda

} // namespace gpu

} // namespace hcp